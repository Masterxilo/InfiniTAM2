#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
using namespace thrust; 
#include <float.h>
#include <array>
#include <gl/freeglut.h>
#define NOMINMAX
#define WINDOWS_LEAN_AND_MEAN
#include <windows.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <lodepng.h>
#include <string> 
#include <vector> 
#include <string>
#include <string>
#include <fstream>
#include <sstream>
#include <streambuf>
#include <iostream>
#include <string>
#include <iterator>
#include <tchar.h>
#pragma comment(lib,"cudart")
#pragma comment(lib,"lodepng")
using namespace std;

#ifndef __HIPCC__
#error This file can only be compiled as a .cu file by nvcc.
#endif

#ifndef __CUDA_ARCH__
#define __CUDACC_RTC__ // HACK calm intellisense about __syncthreads et. al
#endif

#ifndef _WIN64
#error hipMallocManaged and __managed__ require 64 bits. Also, this program is made for windows.
#endif

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 500
#error Always use the latest cuda arch. Old versions dont support any amount of thread blocks being submitted at once.
#endif

#ifdef __CUDA_ARCH__
#define GPU_CODE 1
#else
#define GPU_CODE 0
#endif

#define GPU_ONLY __device__
#define GPU(mem) mem
#define KERNEL __global__ void
#if defined(__HIPCC__) && defined(__CUDA_ARCH__)
#define CPU_AND_GPU __device__
#else
#define CPU_AND_GPU 
#endif























































#ifndef __HIPCC__
// HACK to make intellisense shut up about illegal C++ <<< >>>
#define LAUNCH_KERNEL(kernelFunction, gridDim, blockDim, arguments, ...) ((void)0)
#else
dim3 _lastLaunch_gridDim, _lastLaunch_blockDim;
#define LAUNCH_KERNEL(kernelFunction, gridDim, blockDim, ...) {\
cudaSafeCall(hipGetLastError());\
_lastLaunch_gridDim = dim3(gridDim); _lastLaunch_blockDim = dim3(blockDim);\
kernelFunction << <gridDim, blockDim >> >(__VA_ARGS__);\
cudaSafeCall(hipGetLastError());\
cudaSafeCall(hipDeviceSynchronize()); /* TODO greatly alters the execution order */\
}

#endif






















































#undef assert
#if GPU_CODE
#define assert(x,commentFormat,...) if(!(x)) {printf("%s(%i) : Assertion failed : %s.\n\tblockIdx %d %d %d, threadIdx %d %d %d\n\t<" commentFormat ">\n", __FILE__, __LINE__, #x, xyz(blockIdx), xyz(threadIdx), __VA_ARGS__); *(int*)0 = 0;/* asm("trap;"); illegal instruction*/} 
#else
#define assert(x,commentFormat,...) if(!(x)) {char s[10000]; sprintf_s(s, "%s(%i) : Assertion failed : %s.\n\t<" commentFormat ">\n", __FILE__, __LINE__, #x, __VA_ARGS__); puts(s); flushStd(); DebugBreak(); OutputDebugStringA("! program continues after failed assertion\n\n");} 
#endif

























































// Automatically wrap some functions in cudaSafeCall
#ifdef __HIPCC__ // hack to hide these from intellisense
#define hipDeviceSynchronize(...) cudaSafeCall(hipDeviceSynchronize(__VA_ARGS__))
#define hipMalloc(...) cudaSafeCall(hipMalloc(__VA_ARGS__))
#define hipFree(...) cudaSafeCall(hipFree(__VA_ARGS__))
#define hipMallocManaged(...) cudaSafeCall(hipMallocManaged(__VA_ARGS__))
#endif




















































// cudaSafeCall is an expression that evaluates to 
// 0 when err is hipSuccess (0), such that cudaSafeCall(cudaSafeCall(hipSuccess)) will not block
// this is important because we might have legacy code that explicitly does 
// cudaSafeCall(hipDeviceSynchronize());
// but we extended hipDeviceSynchronize to include this already, giving
// cudaSafeCall(cudaSafeCall(hipDeviceSynchronize()))

// it debug-breaks and returns 
bool cudaSafeCallImpl(hipError_t err, const char * const expr, const char * const file, const int line);

// If err is hipSuccess, cudaSafeCallImpl will return true, early-out of || will make DebugBreak not evaluated.
// The final expression will be 0.
// Otherwise we evaluate debug break, which returns true as well and then return 0.
#define cudaSafeCall(err) \
    !(cudaSafeCallImpl((hipError_t)(err), #err, __FILE__, __LINE__) || ([]() {DebugBreak(); return true;})() )



















































#define xyz(p) p.x, p.y, p.z
#define xy(p) p.x, p.y
#define threadIdx_xyz xyz(threadIdx)























































#define stdouterrfile "stdouterr.txt"
void redirectStd() {
    freopen(stdouterrfile, "w", stdout); // stdout > stdoutfile, 1>stdoutfile
    freopen(stdouterrfile, "w", stderr); // 2>stderrfile
}






















































struct _atinit {
    _atinit() {
        redirectStd();
        // CATCHMALLOCERRORS
        _CrtSetDbgFlag(_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF | _CRTDBG_CHECK_ALWAYS_DF);
    }
} __atinit;






















































struct _atexit {
    ~_atexit() {
        /// Catch remaining cuda errors on shutdown
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());
        cudaSafeCall(hipGetLastError());

        // TODO is this still true?
        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        hipDeviceReset();
    }
} __atexit;






















































int fileExists(TCHAR * file)
{
    WIN32_FIND_DATA FindFileData;
    HANDLE handle = FindFirstFile(file, &FindFileData);
    int found = handle != INVALID_HANDLE_VALUE;
    if (found)
    {
        //FindClose(&handle); this will crash
        FindClose(handle);
    }
    return found;
}






















































std::string readFile(std::string fn) {
    std::ifstream t(fn);
    return std::string(std::istreambuf_iterator<char>(t),
        std::istreambuf_iterator<char>());
}






















































void flushStd() {
    if (!fileExists(stdouterrfile)) return;
    // unlock stdout.txt & stderr
    ::fflush(stdout);
    ::fflush(stderr);
    freopen("CONOUT$", "w", stdout);
    freopen("CONOUT$", "w", stderr);
    std::string s = (
        "<<< " stdouterrfile " >>>\n" + readFile(stdouterrfile)
        );

    OutputDebugStringA(s.c_str());

    remove(stdouterrfile);
    redirectStd();
}






















































/// \returns true if err is hipSuccess
/// Fills errmsg in UNIT_TESTING build.
bool cudaSafeCallImpl(hipError_t err, const char * const expr, const char * const file, const int line)
{
    if (hipSuccess == err) return true;

    char s[10000];
    hipGetLastError(); // Reset error flag
    const char* e = hipGetErrorString(err);
    if (!e) e = "! hipGetErrorString returned 0 !";

    sprintf_s(s, "\n%s(%i) : cudaSafeCall(%s)\nRuntime API error : %s.\n",
        file,
        line,
        expr,
        e);
    puts(s);
    if (err == hipError_t::hipErrorLaunchFailure) {
        printf("maybe illegal memory access, (memcpy(0,0,4) et.al) try the CUDA debugger\n"
            );
    }

    if (err == hipError_t::hipErrorInvalidConfiguration) {
        printf("configuration was (%d,%d,%d), (%d,%d,%d)\n",
            xyz(_lastLaunch_gridDim),
            xyz(_lastLaunch_blockDim)
            );
    }

    if (err == hipError_t::cudaErrorIllegalInstruction) {
        puts("maybe the illegal instruction was asm(trap;) of a failed assertion?");
    }


    flushStd();
    return false;
}






















































/*
* A cut-down local version of gluErrorString to avoid depending on GLU.
*/
const char* fghErrorString(GLenum error)
{
    switch (error) {
    case GL_INVALID_ENUM: return "invalid enumerant";
    case GL_INVALID_VALUE: return "invalid value";
    case GL_INVALID_OPERATION: return "invalid operation";
#ifndef GL_ES_VERSION_2_0
    case GL_STACK_OVERFLOW: return "stack overflow";
    case GL_STACK_UNDERFLOW: return "stack underflow";
#endif
    case GL_OUT_OF_MEMORY: return "out of memory";
    default: return "unknown GL error";
    }
}
























































__managed__ float* sum_Atb; // m x 1
__managed__ float* sum_AtA; // m x m, row major
__managed__ int n;






















































template<typename T, int b>
KERNEL k() {
    assert(false, "problem %d", 5);
}




















































#define REDUCE_BLOCK_SIZE 256
template<class Constructor,int m>
KERNEL constructAndSolve_device(int n) {
    assert(gridDim.y == gridDim.z && gridDim.y == 1);
    assert(blockDim.x == REDUCE_BLOCK_SIZE);
    assert(threadIdx.y == threadIdx.z && threadIdx.y == 0);

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    assert(n == ::n);

    __shared__ float ssum_ai_aiT[REDUCE_BLOCK_SIZE][m][m];// = {0};
    __shared__ float ssum_ai_bi[REDUCE_BLOCK_SIZE][m];// = {0};

    const int tid = threadIdx.x;
    if (i >= n) {

        memset(&ssum_ai_aiT[tid], 0, sizeof(float) * m *m);
        memset(&ssum_ai_bi[tid], 0, sizeof(float) * m);
        return;
    }

    // Computation
    float ai[m]; /* m x 1 */
    float bi;
    Constructor::generate<m>(i, ai, bi);

    // Summands
    float ai_aiT[m][m];
    for (int c = 0; c < m; c++)
        for (int r = 0; r < m; r++) 
            ai_aiT[c][r] = ai[c] * ai[r];

    float ai_bi[m]; /* m x 1 */
    for (int r = 0; r < m; r++)
        ai_bi[r] = ai[r] * bi;

    // -- Summation ---

    // FAST

    // TODO could write to these right away
    memcpy(&ssum_ai_aiT[tid], ai_aiT, sizeof(float) * m *m);
    memcpy(&ssum_ai_bi[tid], ai_bi, sizeof(float) * m);
    __syncthreads();

    // SLOW summation into sum_ai_aiT sum_ai_bi
    __shared__ float sum_ai_aiT[m][m];
    __shared__ float sum_ai_bi[m];
    
    if (tid == 0) {
        memset(&sum_ai_aiT, 0, sizeof(float) * m *m);
        memset(&sum_ai_bi, 0, sizeof(float) * m);

        for (int j = 0; j < REDUCE_BLOCK_SIZE; j++) {
            for (int c = 0; c < m; c++)
                for (int r = 0; r < m; r++)
                    sum_ai_aiT[c][r] += ssum_ai_aiT[j][c][r];

            for (int r = 0; r < m; r++)
                sum_ai_bi[r] += ssum_ai_bi[j][r];
        }
    }
    __syncthreads();

    // FAST summation into sum_ai_aiT[0] sum_ai_bi[0]
    for (int offset = REDUCE_BLOCK_SIZE / 2; offset >= 1; offset /= 2) {
        if (tid >= offset) return;

        for (int c = 0; c < m; c++)
            for (int r = 0; r < m; r++)
                ssum_ai_aiT[tid][c][r] += ssum_ai_aiT[tid + offset][c][r];

        for (int r = 0; r < m; r++)
            ssum_ai_bi[tid][r] += ssum_ai_bi[tid + offset][r];

        __syncthreads();
    }

    // check
    assert(tid == 0);

    if (blockIdx.x == 1) {
        printf("\n");
        for (int r = 0; r < m; r++) {
            for (int c = 0; c < m; c++) {
                printf("%.0f ", sum_ai_aiT[c][r]);
            }
            printf("\n");
        }

        printf("-----\n");
        for (int r = 0; r < m; r++) {
            for (int c = 0; c < m; c++) {
                printf("%.0f ", ssum_ai_aiT[0][c][r]);
            }
            printf("\n");
        }
    }

    for (int c = 0; c < m; c++)
        for (int r = 0; r < m; r++)
            assert(ssum_ai_aiT[0][c][r] == sum_ai_aiT[c][r], "%f %f", ssum_ai_aiT[0][c][r], sum_ai_aiT[c][r]);

    for (int r = 0; r < m; r++)
        assert(ssum_ai_bi[0][r] == sum_ai_bi[r], "%f %f", ssum_ai_bi[0][r], sum_ai_bi[r]);

    // Sum globally
    for (int c = 0; c < m; c++)
        for (int r = 0; r < m; r++)
            atomicAdd(&sum_AtA[r*m + c], ssum_ai_aiT[0][c][r]);

    for (int r = 0; r < m; r++)
        atomicAdd(&sum_Atb[r], ssum_ai_bi[0][r]);
}


















































float assertFinite(float value) {
    assert(_fpclass(value) == _FPCLASS_PD || _fpclass(value) == _FPCLASS_PN || _fpclass(value) == _FPCLASS_PZ ||
        _fpclass(value) == _FPCLASS_ND || _fpclass(value) == _FPCLASS_NN || _fpclass(value) == _FPCLASS_NZ
        , "value = %f is not finite", value);
    return value;
}
















































class Cholesky
{
private:
    std::vector<float> cholesky;
    int size, rank;

public:
    // Solve Ax = b for A symmetric positive-definite of size*size
    static void solve(const float* mat, int size, const float* b, float* result) {
        Cholesky cholA(mat, size);
        cholA.Backsub(result, b);
    }

    /// \f[A = LL*\f]
    /// Produces Cholesky decomposition of the
    /// symmetric, positive-definite matrix mat of dimension size*size
    /// \f$L\f$ is a lower triangular matrix with real and positive diagonal entries
    /// assertFinite is used to detect singular matrices and other non-supported cases.
    Cholesky(const float *mat, int size)
    {
        this->size = size;
        this->cholesky.resize(size*size);

        for (int i = 0; i < size * size; i++) cholesky[i] = assertFinite(mat[i]);

        for (int c = 0; c < size; c++)
        {
            float inv_diag = 1;
            for (int r = c; r < size; r++)
            {
                float val = cholesky[c + r * size];
                for (int c2 = 0; c2 < c; c2++)
                    val -= cholesky[c + c2 * size] * cholesky[c2 + r * size];

                if (r == c)
                {
                    cholesky[c + r * size] = assertFinite(val);
                    if (val == 0) { rank = r; }
                    inv_diag = 1.0f / val;
                }
                else
                {
                    cholesky[r + c * size] = assertFinite(val);
                    cholesky[c + r * size] = assertFinite(val * inv_diag);
                }
            }
        }

        rank = size;
    }

    /// Solves \f[Ax = b\f]
    /// by
    /// * solving Ly = b for y by forward substitution, and then
    /// * solving L*x = y for x by back substitution.
    void Backsub(
        float *x,  //!< out \f$x\f$
        const float *b //!< input \f$b\f$
        ) const
    {
        // Forward
        std::vector<float> y(size);
        for (int i = 0; i < size; i++)
        {
            float val = b[i];
            for (int j = 0; j < i; j++) val -= cholesky[j + i * size] * y[j];
            y[i] = val;
        }

        for (int i = 0; i < size; i++) y[i] /= cholesky[i + i * size];

        // Backward
        for (int i = size - 1; i >= 0; i--)
        {
            float val = y[i];
            for (int j = i + 1; j < size; j++) val -= cholesky[i + j * size] * x[j];
            x[i] = val;
        }
    }
};






















































/** Allocate a block of CUDA memory and memset it to 0 */
template<typename T> static void zeroManagedMalloc(T*& p, const unsigned int count = 1) {
    cudaSafeCall(hipMallocManaged(&p, sizeof(T) * count));
    cudaSafeCall(hipMemset(p, 0, sizeof(T) * count));
}



















































template<class Constructor, int m>
std::array<float, m> constructAndSolve(int n) {
    assert(m < 100);
    ::n = n;

    hipDeviceSynchronize();
    zeroManagedMalloc(sum_AtA, m * m);
    zeroManagedMalloc(sum_Atb, m);
    assert(sum_AtA[0] == 0);
    assert(sum_Atb[0] == 0);
    assert(sum_AtA[m * m-1] == 0);
    assert(sum_Atb[m - 1] == 0);

    LAUNCH_KERNEL(
        (constructAndSolve_device<Constructor, m>),
        ceil(n / (1.f * REDUCE_BLOCK_SIZE)),
        REDUCE_BLOCK_SIZE,
        n);
    hipDeviceSynchronize();

    for (int r = 0; r < m; r++) {
        puts("");
        for (int c = 0; c < m; c++)
            cout << sum_AtA[r * m+ c] << " ";
    }
    assert(sum_AtA[m+1] != 0);
    assert(sum_Atb[m-1] != 0);

    auto x = std::array < float, m >();
    Cholesky::solve(sum_AtA, m, sum_Atb, x.data());
    return x;
}

























































struct ConstructExampleEquation {
    template<int m>
    static GPU_ONLY void generate(const int i, float out_ai[m], float& out_bi/*[1]*/) {
        for (int j = 0; j < m; j++) {
            out_ai[j] = 0;
            if (i == j || i == 0|| j== 0)
                out_ai[j] = 1;
        }
        out_bi = i+1;
    }
};











































void assertApproxEqual(float a, float b, int considered_initial_bits = 20) {
    assert(considered_initial_bits > 8 + 1); // should consider at least sign and full exponent
    assert(considered_initial_bits <= 32);

    unsigned int ai = *(unsigned int*)&a;
    unsigned int bi = *(unsigned int*)&b;
    auto ait = ai >> (32 - considered_initial_bits);
    auto bit = bi >> (32 - considered_initial_bits);

    assert(ait == bit, "%f != %f, %x != %x, %x != %x", 
        a,b,ai,bi,ait,bit
        );
}

struct Trafo {
    CPU_AND_GPU int operator()(int i) {
        return 2 * i;
    }
};
int main(int argc, char** argv)
{
    const int min = 1, max = 6;
    thrust::counting_iterator<int> first(min);
    auto last = first + (max-min) + 1;
    int res = transform_reduce(first, last, Trafo(), 0, thrust::plus<int>());
    assert(res == 2 * (max * (max + 1)) / 2);

    const int m = 6;
    const int n = 2*REDUCE_BLOCK_SIZE;
    auto x = constructAndSolve<ConstructExampleEquation, m>(n);
    float expect[m] = {258.164, -87.2215, -86.2215, -85.2215, -84.2215, -83.2215};
    for (int i = 0; i < m; i++)
        assertApproxEqual(x[i], expect[i]);

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(400, 300);
    glutCreateWindow("Hello World");
    auto e = glGetError();
    glGenTextures(-10,0);
    auto e2 = glGetError(); 
    
    ((void)0);//LAUNCH_KERNEL((k<int, 5>), 1, 1);

    return 0;
}